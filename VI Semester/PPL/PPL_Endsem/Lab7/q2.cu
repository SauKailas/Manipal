#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include "hip/hip_runtime.h"


__global__ void vec_add(int *A, int *B, int *C, int N)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if(tid < N)
		C[tid] = A[tid] + B[tid];
}

int main()
{
	int N;

	printf("Enter the value of N: ");
	scanf("%d", &N);
	int size = N * sizeof(int);

	int *A = (int*)malloc(size);
	int *B = (int*)malloc(size);
	int *C = (int*)malloc(size);

	printf("Enter the elements of A: ");
	for(int i=0; i<N; i++)
		scanf("%d", &A[i]);

	printf("Enter the elements of B: ");
	for(int i=0; i<N; i++)
		scanf("%d", &B[i]);

	int *d_A, *d_B, *d_C;

	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);

	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	dim3 dimGrid(ceil(N/256.0), 1, 1);
	dim3 dimBlock(256, 1, 1);

	vec_add<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);

	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("C : ");
	for(int i=0; i<N; i++)
		printf("%d ", C[i]);
	printf("\n");

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(A);
	free(B);
	free(C);

	return 0;
}