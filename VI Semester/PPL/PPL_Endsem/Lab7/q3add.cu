#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"


__global__ void oddEven(int *A, int N)
{
	int tid = threadIdx.x;
	int idx = 2 * tid + 1;
	if(idx + 1 < N)
	{
		if(A[idx] > A[idx+1])
		{
			int temp = A[idx];
			A[idx] = A[idx+1];
			A[idx+1] = temp;
		}
	}
}

__global__ void evenOdd(int *A, int N)
{
	int tid = threadIdx.x;
	int idx = 2 * tid;
	if(idx + 1 < N)
	{
		if(A[idx] > A[idx+1])
		{
			int temp = A[idx];
			A[idx] = A[idx+1];
			A[idx+1] = temp;
		}
	}
}

int main()
{
	int N;
	int size;

	printf("Enter the number of elements: ");
	scanf("%d", &N);
	size = N * sizeof(int);

	int *A = (int*)malloc(size);

	printf("Enter the array elements: ");
	for(int i=0; i<N; i++)
		scanf("%d", &A[i]);

	int *d_A;

	hipMalloc((void**)&d_A, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

	dim3 dimGrid(1, 1, 1);
	dim3 dimBlock(ceil(N/2), 1, 1);

	for(int i=0; i<N/2; i++)
	{
		oddEven<<<dimGrid, dimBlock>>>(d_A, N);
		evenOdd<<<dimGrid, dimBlock>>>(d_A, N);
	}

	hipMemcpy(A, d_A, size, hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("Sorted Array: ");
	for(int i=0; i<N; i++)
		printf("%d ", A[i]);
	printf("\n");

	hipFree(d_A);
	free(A);

	return 0;
}