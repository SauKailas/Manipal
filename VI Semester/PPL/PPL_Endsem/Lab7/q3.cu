#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void convolution_1d(int *N, int *P, int *M, int width, int mask_width)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int N_start_point = i - (mask_width / 2);
	int Pval = 0,

	for(int j=0; j<mask_width; j++)
		if(N_start_point + j >= 0 && N_start_point + j < width)
			Pval += N[N_start_point + j] * M[j];
	P[i] = Pval;
}

int main()
{
	int width, mask_width;

	printf("Enter the input array width: ");
	scanf("%d", &width);

	printf("Enter the mask width: ");
	scanf("%d", &mask_width);

	int *N = (int*)malloc(width * sizeof(int));
	int *P = (int*)malloc(width * sizeof(int));
	int *M = (int*)malloc(mask_width * sizeof(int));

	printf("Enter the input array elements: ");
	for(int i=0; i<width; i++)
		scanf("%d", &N[i]);

	printf("Enter the mask elements: ");
	for(int i=0; i<mask_width; i++)
		scanf("%d", &M[i]);

	int *d_N, *d_P, *d_P;

	hipMalloc((void**)&d_N, width * sizeof(int));
	hipMalloc((void**)&d_P, width * sizeof(int));
	hipMalloc((void**)&d_M, mask_width * sizeof(int));

	hipMemcpy(d_N, N, width * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_M, M, mask_width * sizeof(int), hipMemcpyHostToDevice);

	convolution_1D<<<1, width>>>(d_N, d_P, d_M, width, mask_width);

	hipMemcpy(P, d_P, width * sizeof(int), hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("P : ");
	for(int i=0; i<width; i++)
		printf("%d ", P[i]);
	printf("\n");

	hipFree(d_N);
	hipFree(d_P);
	hipFree(d_M);
	free(N);
	free(P);
	free(M);

	return 0;
}