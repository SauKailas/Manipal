#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"


__global__ void linear_function(int *X, int *Y, int *Z, int alfa)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int res = X[tid] * alfa + Y[tid];
	Z[tid] = res;
}

int main()
{
	int N, alfa;

	printf("Enter the number of elements: ");
	scanf("%d", &N);
	int size = N * sizeof(int);

	int *X = (int*)malloc(size);
	int *Y = (int*)malloc(size);
	int *Z = (int*)malloc(size);

	printf("Enter the value of alfa: ");
	scanf("%d", &alfa);

	printf("Enter the elements of X: ");
	for(int i=0; i<N; i++)
		scanf("%d", &X[i]);

	printf("Enter the elements of Y: ");
	for(int i=0; i<N; i++)
		scanf("%d", &Y[i]);

	int *d_X, *d_Y, *d_Z;

	hipMalloc((void**)&d_X, size);
	hipMalloc((void**)&d_Y, size);
	hipMalloc((void**)&d_Z, size);

	hipMemcpy(d_X, X, size, hipMemcpyHostToDevice);
	hipMemcpy(d_Y, Y, size, hipMemcpyHostToDevice);

	linear_function<<<1, N>>>(d_X, d_Y, d_Z, alfa);

	hipMemcpy(Z, d_Z, size, hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("Z : ");
	for(int i=0; i<N; i++)
		printf("%d ", Z[i]);
	printf("\n");

	hipFree(d_X);
	hipFree(d_Y);
	hipFree(d_Z);
	free(X);
	free(Y);
	free(Z);

	return 0;
}