#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"


__global__ void row_selection_sort(int *M, int *res, int size)
{
	int rowid = threadIdx.x;
	int colid = threadIdx.y;
	int data = M[rowid * size + colid];
	int pos = 0;

	for(int i=0; i<size; i++)
		if(M[rowid * size + i] < data || (M[rowid * size + i] == data && i < pos))
			pos++;
	res[rowid * size + pos] = data;
}

int main()
{
	int r, c;

	printf("Enter the number of rows and columns: ");
	scanf("%d %d", &r, &c);
	int size = r * c * sizeof(int);

	int *M = (int*)malloc(size);
	int *res = (int*)malloc(size);

	printf("Enter the matrix elements: ");
	for(int i=0; i<(r*c); i++)
		scanf("%d", &M[i]);

	int *d_M, *d_res;

	hipMalloc((void**)&d_M, size);
	hipMalloc((void**)&d_res, size);

	hipMemcpy(d_M, M, size, hipMemcpyHostToDevice);

	row_selection_sort<<<1, (r, c, 1)>>>(d_M, d_res, c);

	hipMemcpy(res, d_res, size, hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("Resultant Matrix:\n");
	for(int i=0; i<(r*c); i++)
	{
		if(i % c == 0)
			printf("\n");
		printf("%d ", res[i]);
	}

	hipFree(d_M);
	hipFree(d_res);
	free(M);
	free(res);

	return 0;
}