#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"


__global__ void vec_add(int *A, int *B, int *C)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	C[tid] = A[tid] + B[tid];
}

int main()
{
	int N;

	printf("Enter the value of N: ");
	scanf("%d", &N);
	int size = N * sizeof(int);

	int *A = (int*)malloc(N * sizeof(size));
	int *B = (int*)malloc(N * sizeof(size));
	int *C = (int*)malloc(N * sizeof(size));

	printf("Enter the elements of A: ");
	for(int i=0; i<N; i++)
		scanf("%d", &A[i]);

	printf("Enter the elements of B: ");
	for(int i=0; i<N; i++)
		scanf("%d", &B[i]);

	int *d_A, *d_B, *d_C;

	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);

	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	vec_add<<<1, N>>>(d_A, d_B, d_C);

	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("C : ");
	for(int i=0; i<N; i++)
		printf("%d ", C[i]);
	printf("\n");

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(A);
	free(B);
	free(C);

	return 0;
}