#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void sin_rad(float *A, float *B)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	float val = A[tid];
	float res = sin(val);
	B[tid] = res;x
}

int main()
{
	int N;

	printf("Enter the value of N: ");
	scanf("%d", &N);
	int size = N * sizeof(float);

	float *A = (float*)malloc(size);
	float *B = (float*)malloc(size);

	printf("Enter the elements of A: ");
	for(int i=0; i<N; i++)
		scanf("%f", &A[i]);

	float *d_A, *d_B;

	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);

	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

	sin_rad<<<1, N>>>(d_A, d_B);

	hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("B : ");
	for(int i=0; i<width; i++)
		printf("%.3f ", B[i]);
	printf("\n");

	hipFree(d_A);
	hipFree(d_B);
	free(A);
	free(B);

	return 0;
}