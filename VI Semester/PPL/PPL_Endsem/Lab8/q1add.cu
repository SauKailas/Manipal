#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void Reverse_Words(char *str, int word_len)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	char temp;

	for(int i=0; i<(word_len/2); i++)
	{
		temp = str[tid + i];
		str[tid + i] = str[tid + word_len - i - 1];
		str[tid + word_len - i + 1] = temp;
	}
}

int main()
{
	int N, word_len;
	char S[100], str[100];
	int len = 0;

	printf("Enter the value of N: ");
	scanf("%d", &N);

	printf("Enter the length of each word: ");
	scanf("%d", &word_len);

	printf("Enter the string: ");
	gets(S);

	for(int i=0; i<strlen(S); i++)
	{
		if(S[i] == ' ')
			continue;
		str[len++] = S[i];
	}
	len--;

	char *d_str

	hipMalloc((void**)&d_str, len * sizeof(char));

	hipMemcpy(d_str, str, len * sizeof(char), hipMemcpyHostToDevice);

	Reverse_Words<<<1, N>>>(d_str, word_len);

	hipMemcpy(str, d_str, len * sizeof(char), hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("Reversed Words: ");
	for(int i=0; i<len; i++)
	{
		if(i % word_len == 0)
			printf(" ");
		printf("%c", str[i]);
	}

	hipFree(d_str);

	return 0;
}