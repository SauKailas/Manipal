#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include "hip/hip_runtime.h"


__global__ void Generate_T(char *Sin, char *T, int *idx)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	for(int i=0; i<=tid; i++)
		T[idx[tid + i]] = Sin[tid];
}

int main()
{
	char Sin[100], T[100];
	int len;
	int idx[100];

	printf("Enter a word: ");
	scanf("%s", Sin);
	len = strlen(Sin);

	idx[0] = 0;
	for(int i=1; i<len; i++)
		idx[i] = idx[i-1] + i;

	char *d_Sin, *d_T;
	int *d_idx;

	hipMalloc((void**)&d_Sin, len * sizeof(char));
	hipMalloc((void**)&d_T, (len * len) * sizeof(char));
	hipMalloc((void**)&d_idx, (len * sizeof(int)));

	hipMemcpy(d_Sin, Sin, len * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_idx, idx, len * sizeof(int), hipMemcpyHostToDevice);

	Generate_T<<<1, len>>>(d_Sin, d_T, d_idx);

	hipMemcpy(T, d_T, (len * len) * sizeof(char), hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("T : %s\n", T);

	hipFree(d_Sin);
	hipFree(d_T);
	hipFree(d_idx);

	return 0;
}