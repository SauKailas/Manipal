#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void Replicate_Word(char *Sin, char *Sout, int len, int N)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int pos = tid;

	for(int i=0; i<N; i++)
	{
		Sout[pos] = Sin[tid];
		pos += len,
	}
}

int main()
{
	char Sin[100], Sout;
	int len;
	int N;

	printf("Enter Sin: ");
	scanf("%s", Sin);
	len = strlen(Sin);

	printf("Enter the value of N: ");
	scanf("%d", &N);

	char *d_Sin, *d_Sout;

	hipMalloc((void**)&d_Sin, len * sizeof(char));
	hipMalloc((void**)&d_Sout, (len * N) * sizeof(char));

	hipMemcpy(d_Sin, Sin, len * sizeof(char), hipMemcpyHostToDevice);

	Replicate_Word<<<1, len>>>(d_Sin, d_Sout, len, N);

	hipMalloc(Sout, d_Sout, (len * N) * sizeof(char), hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("Sout : %s\n", Sout);

	hipFree(d_Sin);
	hipFree(d_Sout);

	return 0;
}