#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void Count_Word_Occurrences(char *str, char *word, int str_len, int word_len, int count)
{
	int tid = threadIdx.x;

	while(tid < str_len)
	{
		int i = 0;
		while(i < word_len && str[tid + i] == word[i])
			i++;

		if(i == word_len)
		{
			atomicAdd(count, 1);
			tid += word_len;
		}
		else
			tid++;
	}
}

int main()
{
	int str_len, word_len;

	printf("Enter string length: ");
	scanf("%d", &str_len);

	printf("Enter word length: ");
	scanf("%d", &word_len);

	char *str = (char*)malloc(str_len * sizeof(char));
	char *word = (char*)malloc(word_len * sizeof(char));

	printf("Enter the string: ");
	gets(str);

	printf("Enter the word: ");
	gets(word);

	char *d_word, *d_str;
	int *count, *d_count;

	hipMalloc((void**)&d_str, str_len * sizeof(char));
	hipMalloc((void**)&d_word, word_len * sizeof(char));
	hipMalloc((void**)&d_count, sizeof(int));

	hipMemcpy(d_str, str, str_len * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_word, word, word_len * sizeof(char), hipMemcpyHostToDevice);
	hipMemset(d_count, 0, sizeof(int));

	Count_Word_Occurrences<<<1, 1>>>(d_str, d_word, str_len, word_len, d_count);

	hipMemcpy(count, d_count, sizeof(int), hipMemcpyDeviceToHost);

	printf("RESULTS:\n");
	printf("Number of Occurrences: %d\n", count);

	hipFree(d_str);
	hipFree(d_word);
	hipFree(d_count);
	free(str);
	free(word);
	free(count);

	return 0;
}