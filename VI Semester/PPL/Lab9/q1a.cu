#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"


__global__ void row_add(int *M1, int *M2, int *res, int c)
{
	int tid = threadIdx.x;
	for(int i=0; i<c; i++)
		res[tid * c + i] = M1[tid * c + i] + M2[tid * c + i];
}

int main()
{
	int r, c;

	printf("Enter the number of rows: ");
	scanf("%d", &r);
	printf("Enter the number of columns: ");
	scanf("%d", &c);

	int size = r * c * sizeof(int);

	int *M1 = (int*)malloc(size);
	int *M2 = (int*)malloc(size);
	int *res = (int*)malloc(size);

	printf("Enter the elements of matrix 1: ");
	for(int i=0; i<(r*c); i++)
		scanf("%d", &M1[i]);

	printf("Enter the elements of matrix 2: ");
	for(int i=0; i<(r*c); i++)
		scanf("%d", &M2[i]);

	printf("Entered Matrix 1:\n");
	for(int i=0; i<r; i++)
	{
		for(int j=0; j<c; j++)
			printf("%d ", M1[i*c+j]);
		printf("\n");
	}

	printf("\nEntered Matrix 2:\n");
	for(int i=0; i<r; i++)
	{
		for(int j=0; j<c; j++)
			printf("%d ", M2[i*c+j]);
		printf("\n");
	}

	int *d_M1, *d_M2, *d_res;

	hipMalloc(&d_M1, size);
	hipMalloc(&d_M2, size);
	hipMalloc(&d_res, size);

	hipMemcpy(d_M1, M1, size, hipMemcpyHostToDevice);
	hipMemcpy(d_M2, M2, size, hipMemcpyHostToDevice);

	row_add<<<1, r>>>(d_M1, d_M2, d_res, c);

	hipMemcpy(res, d_res, size, hipMemcpyDeviceToHost);

	printf("\nResultant Matrix:\n");
	for(int i=0; i<r; i++)
	{
		for(int j=0; j<c; j++)
			printf("%d ", res[i*c+j]);
		printf("\n");
	}

	hipFree(d_M1);
	hipFree(d_M2);
	hipFree(d_res);

	return 0;
}