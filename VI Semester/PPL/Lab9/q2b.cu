#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"


__global__ void matrix_mult_colwise(int *A, int *B, int *C, int ha, int hb)
{
	int colIdB = blockIdx.x * blockDim.x + threadIdx.x;
	int wb = blockDim.x;
	int sum;

	for(int rowIdA = 0; rowIdA < ha; rowIdA++)
	{
		sum = 0;
		for(int k = 0; k < hb; k++)
			sum += A[rowIdA * hb + k] * B[k * wb + colIdB];
		C[rowIdA * wb + colIdB] = sum;
	}
}

int main()
{
	int wa, wb, ha, hb;

	printf("Enter the dimensions of matrix A: ");
	scanf("%d %d", &ha, &wa);

	printf("Enter the dimensions of matrix B: ");
	scanf("%d %d", &hb, &wb);

	int sizeA = wa * ha * sizeof(int);
	int sizeB = wb * hb * sizeof(int);
	int sizeC = ha * wb * sizeof(int);

	int *A = (int*)malloc(sizeA);
	int *B = (int*)malloc(sizeB);
	int *C = (int*)malloc(sizeC);

	printf("Enter the elements of matrix A: ");
	for(int i=0; i<(wa*ha); i++)
		scanf("%d", &A[i]);

	printf("Enter the elements of matrix B: ");
	for(int i=0; i<(wb*hb); i++)
		scanf("%d", &B[i]);

	int *d_A, *d_B, *d_C;

	hipMalloc((void**)&d_A, sizeA);
	hipMalloc((void**)&d_B, sizeB);
	hipMalloc((void**)&d_C, sizeB);

	hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);

	dim3 dimGrid(1, 1, 1);
	dim3 dimBlock(wb, 1, 1);
	matrix_mult_colwise<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, ha, hb);

	hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);

	printf("Resultant Matrix:\n");
	for(int i=0; i<ha; i++)
	{
		for(int j=0; j<wb; j++)
			printf("%d ", C[i]);
		printf("\n");
	}

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;
}