#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"


__global__ void ele_add(int *M1, int *M2, int *res)
{
	int tid = threadIdx.x;
	res[tid] = M1[tid] + M2[tid];
}

int main()
{
	int r, c;

	printf("Enter the number of rows: ");
	scanf("%d", &r);
	printf("Enter the number of columns: ");
	scanf("%d", &c);

	int size = r * c * sizeof(int);

	int *M1 = (int*)malloc(size);
	int *M2 = (int*)malloc(size);
	int *res = (int*)malloc(size);

	printf("Enter the elements of matrix 1: ");
	for(int i=0; i<(r*c); i++)
		scanf("%d", &M1[i]);

	printf("Enter the elements of matrix 2: ");
	for(int i=0; i<(r*c); i++)
		scanf("%d", &M2[i]);

	printf("Entered Matrix 1:\n");
	for(int i=0; i<r; i++)
	{
		for(int j=0; j<c; j++)
			printf("%d ", M1[i*c+j]);
		printf("\n");
	}

	printf("\nEntered Matrix 2:\n");
	for(int i=0; i<r; i++)
	{
		for(int j=0; j<c; j++)
			printf("%d ", M2[i*c+j]);
		printf("\n");
	}

	int *d_M1, *d_M2, *d_res;

	hipMalloc(&d_M1, size);
	hipMalloc(&d_M2, size);
	hipMalloc(&d_res, size);

	hipMemcpy(d_M1, M1, size, hipMemcpyHostToDevice);
	hipMemcpy(d_M2, M2, size, hipMemcpyHostToDevice);

	ele_add<<<1, r*c>>>(d_M1, d_M2, d_res);

	hipMemcpy(res, d_res, size, hipMemcpyDeviceToHost);

	printf("\nResultant Matrix:\n");
	for(int i=0; i<r; i++)
	{
		for(int j=0; j<c; j++)
			printf("%d ", res[i*c+j]);
		printf("\n");
	}

	hipFree(d_M1);
	hipFree(d_M2);
	hipFree(d_res);

	return 0;
}