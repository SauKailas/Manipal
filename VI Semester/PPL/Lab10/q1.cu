#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"


__global__ void SpVM_CSR(int num_rows, int *data, int *col_index, int *row_ptr, int *X, int *Y)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;

	if(row < num_rows)
	{
		int dot = 0;
		int row_start = row_ptr[row];
		int row_end = row_ptr[row+1];
		for(int elem = row_start; elem < row_end; elem++)
			dot += data[elem] * X[col_index[elem]];
		Y[row] = dot;
	}
}

int main()
{
	int r, c;

	printf("Enter the number of rows: ");
	scanf("%d", &r);
	printf("Enter the number of columns: ");
	scanf("%d", &c);

	int **A = (int**)malloc(r * sizeof(int*));
	for(int i=0; i<r; i++)
		A[i] = (int*)malloc(c * sizeof(int));

	printf("Enter the matrix elements:\n");
	for(int i=0; i<r; i++)
		for(int j=0; j<c; j++)
			scanf("%d", &A[i][j]);

	int *X = (int*)malloc(c * sizeof(int));
	int *Y = (int*)malloc(r * sizeof(int));

	printf("Enter the elements of vector X: ");
	for(int i=0; i<c; i++)
		scanf("%d", &X[i]);

	int data[20], col_index[20], row_ptr[20];
	row_ptr[0] = 0;
	int count = 0;

	for(int i=0; i<r; i++)
	{
		int c = 0;
		for(int j=0; j<c; j++)
		{
			if(A[i][j] != 0)
			{
				c++;
				data[count] = A[i][j];
				col_index[count] = j;
				count++;
			}
		}
		row_ptr[i+1] = row_ptr[i] + c;
	}

	int *d_data, *d_col_index, *d_row_ptr, *d_X, *d_Y;
	hipMalloc((void**)&d_data, count * sizeof(int));
	hipMalloc((void**)&d_col_index, count * sizeof(int));
	hipMalloc((void**)&d_row_ptr, (r + 1) * sizeof(int));
	hipMalloc((void**)&d_X, c * sizeof(int));
	hipMalloc((void**)&d_Y, r * sizeof(int));

	hipMemcpy(d_data, data, count * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_col_index, col_index, count * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_row_ptr, row_ptr, (r + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_X, X, c * sizeof(int), hipMemcpyHostToDevice);

	SpVM_CSR<<<1, r>>>(r, d_data, d_col_index, d_row_ptr, d_X, d_Y);

	hipMemcpy(Y, d_Y, r * sizeof(int), hipMemcpyDeviceToHost);

	printf("Resultant Vector Y: ");
	for(int i=0; i<r; i++)
		printf("%d ", Y[i]);
	printf("\n");

	hipFree(d_data);
	hipFree(d_col_index);
	hipFree(d_row_ptr);
	hipFree(d_X);
	hipFree(d_Y);

	return 0;
}