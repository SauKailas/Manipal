#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"


__global__ void Replace(int *A, int *B, int N)
{
	int tid = blockIdx.x* blockDim.x + threadIdx.x;

	for(int i=0; i<N; i++)
	{
		int ele = A[tid * N + i];
		int temp = ele;
		for(int j=0; j<tid; j++)
			ele *= temp;
		B[tid * N + i] = ele;
	}
}

int main()
{
	int M, N;

	printf("Enter the value of M: ");
	scanf("%d", &M);

	printf("Enter the value of N: ");
	scanf("%d", &N);

	int size = M * N * sizeof(int);

	int *A = (int*)malloc(size);
	int *B = (int*)malloc(size);

	printf("Enter the matrix elements:\n");
	for(int i=0; i<(M * N); i++)
		scanf("%d", &A[i]);

	printf("Entered Matrix:\n");
	for(int i=0; i<M; i++)
	{
		for(int j=0; j<N; j++)
			printf("%d ", A[i * N + j]);
		printf("\n");
	}

	int *d_A, *d_B;

	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);

	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

	Replace<<<1, M>>>(d_A, d_B, N);

	hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);

	printf("Resultant Matrix:\n");
	for(int i=0; i<M; i++)
	{
		for(int j=0; j<N; j++)
			printf("%d ", B[i * N + j]);
		printf("\n");
	}

	hipFree(d_A);
	hipFree(d_B);

	return 0;
}