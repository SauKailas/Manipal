#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"


__global__ void matrixMultiply(int *a, int *b, int *c, int rows_a, int cols_a, int cols_b) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows_a && col < cols_b) {
        int sum = 0;
        for (int k = 0; k < cols_a; ++k) {
            sum += a[row * cols_a + k] * b[k * cols_b + col];
        }
        c[row * cols_b + col] = sum;
    }
}

int main() {
    int rows_a, cols_a, rows_b, cols_b;

    printf("Enter the number of rows and columns for matrix A: ");
    scanf("%d %d", &rows_a, &cols_a);

    printf("Enter the number of rows and columns for matrix B: ");
    scanf("%d %d", &rows_b, &cols_b);

    if (cols_a != rows_b) {
        printf("Matrix multiplication not possible. Number of columns in A must be equal to the number of rows in B.\n");
        return 1;
    }

    int *a, *b, *c; 
    int *d_a, *d_b, *d_c; 
    int size_a = rows_a * cols_a * sizeof(int);
    int size_b = rows_b * cols_b * sizeof(int);
    int size_c = rows_a * cols_b * sizeof(int);

    a = (int*)malloc(size_a);
    b = (int*)malloc(size_b);
    c = (int*)malloc(size_c);

    printf("Enter the elements for matrix A (%d x %d):\n", rows_a, cols_a);
    for (int i = 0; i < rows_a; ++i) {
        for (int j = 0; j < cols_a; ++j) {
            scanf("%d", &a[i * cols_a + j]);
        }
    }

    printf("Enter the elements for matrix B (%d x %d):\n", rows_b, cols_b);
    for (int i = 0; i < rows_b; ++i) {
        for (int j = 0; j < cols_b; ++j) {
            scanf("%d", &b[i * cols_b + j]);
        }
    }

    hipMalloc((void**)&d_a, size_a);
    hipMalloc((void**)&d_b, size_b);
    hipMalloc((void**)&d_c, size_c);

    hipMemcpy(d_a, a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size_b, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16); 
    dim3 dimGrid(ceil((float)cols_b / dimBlock.x), ceil((float)rows_a / dimBlock.y));


    matrixMultiply<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, rows_a, cols_a, cols_b);

    hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost);

    printf("Result Matrix:\n");
    for (int i = 0; i < rows_a; ++i) {
        for (int j = 0; j < cols_b; ++j) {
            printf("%d ", c[i * cols_b + j]);
        }
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(a);
    free(b);
    free(c);

    return 0;
}
